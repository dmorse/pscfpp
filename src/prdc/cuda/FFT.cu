/*
* PSCF - Polymer Self-Consistent Field 
*
* Copyright 2015 - 2025, The Regents of the University of Minnesota
* Distributed under the terms of the GNU General Public License.
*/

#include "FFT.tpp"

namespace Pscf {
namespace Prdc {
namespace Cuda {

   using namespace Util;

   // Forward real-to-complex transform, explicit specializations.
   template<>
   void FFT<1>::makePlans()
   {
      int n0 = meshDimensions_[0];
      #ifdef SINGLE_PRECISION
      hipfftPlan1d(&rcfPlan_, n0, HIPFFT_R2C, 1);
      hipfftPlan1d(&criPlan_, n0, HIPFFT_C2R, 1);
      hipfftPlan1d(&ccPlan_, n0, HIPFFT_C2C, 1);
      #else
      hipfftPlan1d(&rcfPlan_, n0, HIPFFT_D2Z, 1);
      hipfftPlan1d(&criPlan_, n0, HIPFFT_Z2D, 1);
      hipfftPlan1d(&ccPlan_, n0, HIPFFT_Z2Z, 1);
      #endif
   }

   template <>
   void FFT<2>::makePlans()
   {
      int n0 = meshDimensions_[0];
      int n1 = meshDimensions_[1];
      #ifdef SINGLE_PRECISION
      hipfftPlan2d(&rcfPlan_, n0, n1, HIPFFT_R2C);
      hipfftPlan2d(&criPlan_, n0, n1, HIPFFT_C2R);
      hipfftPlan2d(&ccPlan_, n0, n1, HIPFFT_C2C);
      #else
      hipfftPlan2d(&rcfPlan_, n0, n1, HIPFFT_D2Z);
      hipfftPlan2d(&criPlan_, n0, n1, HIPFFT_Z2D);
      hipfftPlan2d(&ccPlan_, n0, n1, HIPFFT_Z2Z);
      #endif
   }

   template <>
   void FFT<3>::makePlans()
   {
      int n0 = meshDimensions_[0];
      int n1 = meshDimensions_[1];
      int n2 = meshDimensions_[2];
      #ifdef SINGLE_PRECISION
      hipfftPlan3d(&rcfPlan_, n0, n1, n2, HIPFFT_R2C);
      hipfftPlan3d(&criPlan_, n0, n1, n2, HIPFFT_C2R);
      hipfftPlan3d(&ccPlan_, n0, n1, n2, HIPFFT_C2C);
      #else
      hipfftPlan3d(&rcfPlan_, n0, n1, n2, HIPFFT_D2Z);
      hipfftPlan3d(&criPlan_, n0, n1, n2, HIPFFT_Z2D);
      hipfftPlan3d(&ccPlan_, n0, n1, n2, HIPFFT_Z2Z);
      #endif
   }

   // Explicit instantiation of relevant class instances
   template class FFT<1>;
   template class FFT<2>;
   template class FFT<3>;

}
}
}
