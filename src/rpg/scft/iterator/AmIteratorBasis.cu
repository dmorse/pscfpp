/*
* PSCF - Polymer Self-Consistent Field
*
* Copyright 2015 - 2025, The Regents of the University of Minnesota
* Distributed under the terms of the GNU General Public License.
*/

#include "AmIteratorBasis.tpp"

namespace Pscf {
namespace Rpg {
   template class AmIteratorBasis<1>;
   template class AmIteratorBasis<2>;
   template class AmIteratorBasis<3>;
}
}
